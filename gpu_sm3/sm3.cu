#include "hip/hip_runtime.h"
/**
  Copyright © 2017 Odzhan. All Rights Reserved.

  Redistribution and use in source and binary forms, with or without
  modification, are permitted provided that the following conditions are
  met:

  1. Redistributions of source code must retain the above copyright
  notice, this list of conditions and the following disclaimer.

  2. Redistributions in binary form must reproduce the above copyright
  notice, this list of conditions and the following disclaimer in the
  documentation and/or other materials provided with the distribution.

  3. The name of the author may not be used to endorse or promote products
  derived from this software without specific prior written permission.

  THIS SOFTWARE IS PROVIDED BY AUTHORS "AS IS" AND ANY EXPRESS OR
  IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
  WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
  DISCLAIMED. IN NO EVENT SHALL THE AUTHOR BE LIABLE FOR ANY DIRECT,
  INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
  (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
  SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION)
  HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT,
  STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN
  ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
  POSSIBILITY OF SUCH DAMAGE. */

#include "sm3.h"
#include <string.h>
#include "stdio.h"
#include <time.h>

#define BLOCK_SIZE 256

#define F1(x,y,z)(((x)^(y)^(z)))
#define FF(x,y,z)(((x)&(y))^((x)&(z))^((y)&(z))) 
#define GG(x,y,z)(((x)&(y))^(~(x)&(z)))

#define P0(x)x^R(x,9)^R(x,17)
#define P1(x)x^R(x,15)^R(x,23)

__device__
unsigned int bswap32(unsigned int x){
  unsigned int result;
  unsigned char* out =(unsigned char *)&result;
  unsigned char* input = (unsigned char *)&x;
  out[3] = input[0];
  out[2] = input[1];
  out[1] = input[2];
  out[0] = input[3];
  return result;
}

__device__
unsigned long bswap64(unsigned long x){
  unsigned long result;
  unsigned char* out = (unsigned char *)&result;
  unsigned char* input = (unsigned char *)&x;

  out[7] = input[0];
  out[6] = input[1];
  out[5] = input[2];
  out[4] = input[3];
  out[3] = input[4];
  out[2] = input[5];
  out[1] = input[6];
  out[0] = input[7];
  return result;
}

__device__

W R_replace(W v,int n){

  
  int right_shift = 32 - n;
  n = n % 32;
  right_shift = (right_shift + 32) % 32;
  
  W r0 = (v) << (n);
  W r1 = (v) >> right_shift;
  W r = r0 | r1;
  printf("v %u, n %d, right_shift %d\n", v,n, right_shift);
  printf("r0 %u, r1 %u, r %u\n", r0, r1, r);
  return r;
}

__device__

W R(W v,int n){

  int right_shift = 32 - n;
  n = n % 32;
  right_shift = (right_shift + 32) % 32;
  
  W r0 = (v) << (n);
  W r1 = (v) >> right_shift;
  W r = r0 | r1;
  
  return r;
}
__device__ 
void sm3_compress(sm3_ctx *c) {
    W t1,t2,i,t,s1,s2,x[8],w[68];

    // load data
    F(16)w[i]=rev32(c->x.w[i]);

    for(i=16;i<68;i++)
      w[i]=P1(w[i-16]^w[i-9]^R(w[i-3],15))^R(w[i-13],7)^w[i- 6];

    // load internal state
    F(8)x[i]=c->s[i];
    
    // compress data
    F(64) {
      t=(i<16)?0x79cc4519:0x7a879d8a;         
      s2=R(x[0],12);      
      s1=R(s2+x[4]+R(t,i),7);
      s2^=s1;
      if(i<16) {
        t1=F1(x[0],x[1],x[2])+x[3]+s2+(w[i]^w[i+4]);
        t2=F1(x[4],x[5],x[6])+x[7]+s1+w[i];
      } else {
        t1=FF(x[0],x[1],x[2])+x[3]+s2+(w[i]^w[i+4]);
        t2=GG(x[4],x[5],x[6])+x[7]+s1+w[i];      
      }
      x[3]=x[2];x[2]=R(x[1],9);x[1]=x[0];x[0]=t1;
      x[7]=x[6];x[6]=R(x[5],19);x[5]=x[4];x[4]=P0(t2);     
    }

    F(8)c->s[i]^=x[i];
}

__device__ 
void sm3_init(sm3_ctx *c) {    
    c->s[0]=0x7380166f;
    c->s[1]=0x4914b2b9;
    c->s[2]=0x172442d7;
    c->s[3]=0xda8a0600;
    c->s[4]=0xa96f30bc;
    c->s[5]=0x163138aa;
    c->s[6]=0xe38dee4d;
    c->s[7]=0xb0fb0e4e;
    c->len =0;
}

__device__ 
void sm3_update(sm3_ctx *c, const void *in, W len) {
    B *p=(B*)in;
    W i, idx;
    
    // index = len % 64
    idx = c->len & 63;
    // update total length
    c->len += len;
    
    for (i=0;i<len;i++) {
      // add byte to buffer
      c->x.b[idx]=p[i]; idx++;
      // buffer filled?
      if(idx==64) {
        // compress it
        sm3_compress(c);
        idx=0;
      }
    }
}

__device__ 
void sm3_final(void *out, sm3_ctx *c) {
    W i,len,*p=(W*)out;
    
    // get index
    i = len = c->len & 63;
    // zero remainder of buffer
    while(i < 64) c->x.b[i++]=0;
    // add 1 bit
    c->x.b[len]=0x80;
    
    // exceeds or equals area for total bits?
    if(len >= 56) {
      // compress it
      sm3_compress(c);
      // zero buffer
      F(16)c->x.w[i]=0;
    }
    // add total length in bits
    c->x.q[7]=rev64(c->len*8);
    // compress it
    sm3_compress(c);
    // return hash
    F(8)p[i]=rev32(c->s[i]);
}


//compare 256bit integer (32 byte)
__device__
int isLessThan(const unsigned char* a, const unsigned char* b, int num_bytes){      
  for( int i = 0; i < num_bytes; i++){
    if(*(a+i) < *(b+i))
      return 1;
    else if (*(a+i) > *(b+i))
      return 0;
    else
      continue;
  }
  return 0;
}

__device__
void hash2(const char* a, int msg_len,int b, unsigned char* out){  
  int len_a = msg_len;
  int len_b = sizeof(b);
  sm3_ctx c;  
  sm3_init(&c);
  sm3_update(&c, a, len_a);
  sm3_update(&c, &b, len_b);  
  sm3_final(out, &c);
}


__global__ 
void gpu_sm3(void* d_m, int len_m ,unsigned char* d_h){
  int i = blockIdx.x*blockDim.x + threadIdx.x;

  if (i == 0){
    sm3_ctx c;
    sm3_init(&c);    
    sm3_update(&c, d_m, len_m);      
    sm3_final(d_h, &c);    
  }
}


__device__ bool found_flag = false;
__device__ int valid_nounce = -1;



__global__
void find_valid_nounce(int n, const unsigned char* boundry, const char* msg, int msg_len){
  int nounce = blockIdx.x*blockDim.x + threadIdx.x;
  //printf("nounce is %d\n", nounce);
  if(nounce < n){
    if(!found_flag){
      //int valid =  dev_valid_nounce(nounce, boundry, msg, msg_len);    
      unsigned char hash_out[32];  
      hash2(msg, msg_len, nounce, hash_out);
      int valid = isLessThan(hash_out, boundry, 32);  
      if(valid){      
        found_flag = true;
        valid_nounce = nounce;
      }
      __threadfence();
    }    
  }
}

void host_find_valid_nounce(Q N, const unsigned char* boundry, const char* msg){
  
  int msg_len = strlen(msg);  

  clock_t begin = clock();
  char* d_msg;
  unsigned char *d_boundry;
  
  volatile bool found = false;

  int error;

  error = hipMalloc(&d_boundry, 32*sizeof(char));   
  error = hipMalloc(&d_msg, strlen(msg)*sizeof(unsigned char)); 

  int memcpyStatus;  
  memcpyStatus =hipMemcpy(d_boundry, boundry, 32*sizeof(unsigned char), hipMemcpyHostToDevice);  
  memcpyStatus =hipMemcpy(d_msg, msg,strlen(msg)*sizeof(char), hipMemcpyHostToDevice);
  
  
  find_valid_nounce<<<(N+BLOCK_SIZE-1)/BLOCK_SIZE, BLOCK_SIZE>>>(N, d_boundry, d_msg,  msg_len);
  
  int answer;
  hipMemcpyFromSymbol(&answer, HIP_SYMBOL(valid_nounce), sizeof(answer), 0, hipMemcpyDeviceToHost);
  clock_t end = clock();
  double time_spent = (double)(end - begin) / CLOCKS_PER_SEC;  
  if (answer == -1){
    printf("cannot find valid nounce for the message %s\n", msg);
  }
  else
    printf("The valid nounce for %s is %d\n", msg, answer);
  printf("time spentd is %f\n", time_spent);
}



// int main(){
//   cpu_find_valid_nounce();
//   return 0;
//   // test_gpu_sm3_working_fine();
//   // return 0;  
// }